#include "hip/hip_runtime.h"
#include "matrix_viz.h"

#include <stdio.h>

namespace dart {

// -=-=-=-=-=-=-=-=-=- kernels -=-=-=-=-=-=-=-=-=-
__global__ void gpu_visualizeMatrix(
     const float * mxData,
     const int mxCols,
     const int mxRows,
     uchar3 * img,
     const int width,
     const int height,
     const uchar3 zeroColor,
     const float minVal,
     const float maxVal) {

    const int x = blockIdx.x*blockDim.x + threadIdx.x;
    const int y = blockIdx.y*blockDim.y + threadIdx.y;

    if (x >= width || y >= height) {
        return;
    }

    int mxCol = (float)mxCols*x/width;
    int mxRow = (float)mxRows*y/height;

    mxCol = min(max(0,mxCol),mxCols-1);
    mxRow = min(max(0,mxRow),mxRows-1);

//    printf("%d,%d\n",mxRow,mxCol);
    const float val = mxData[mxRow*mxCols + mxCol];
    //const int val = 0.0f;

    if (val == 0.0f) {
        img[x + y*width] = zeroColor;
        return;
    }

    float a = min(max(0.0f,(val - minVal)/(val - maxVal)),1.0f);
    img[x + y*width] = make_uchar3(255*a,255*a,255*a);

}

// -=-=-=-=-=-=-=-=-=- interface -=-=-=-=-=-=-=-=-=-
void visualizeMatrix(const float * mxData,
     const int mxCols,
     const int mxRows,
     uchar3 * img,
     const int width,
     const int height,
     const uchar3 zeroColor,
     const float minVal,
     const float maxVal) {

    dim3 block(16,8,1);
    dim3 grid( ceil( width / (float)block.x), ceil(height / (float)block.y ));

    gpu_visualizeMatrix<<<grid,block>>>(mxData,mxCols,mxRows,img,width,height,zeroColor,minVal,maxVal);

}

}
